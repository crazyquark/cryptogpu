#include <cutil_inline.h>		  // Utilitare CUDA
#include "CUDAEncrypter.h"
#include "Log.h"
#include "Exceptions.h"
#include "CUDAEncrypterKernel.cu" // Cod ce ruleaza pe GPU
/**
* Contine functiile de criptare folosind CUDA,
* partea de cod ce ruleaza pe gazda
*/

extern "C"
{
static unsigned int timer = 0;

/**
* Initializare CUDA
*/
void CUDAInit()
{
	int deviceCount;
    hipGetDeviceCount(&deviceCount);
    if(deviceCount == 0)
	{
		LogError("No CUDA device available!");
		return;
	}

	// Setam device-ul CUDA(GPU-ul cel mai rapid din sistem)
	int cudaDevice = cutGetMaxGflopsDeviceId();
	hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, cudaDevice);
	LogMessage("Using device: %s", deviceProp.name);
	hipSetDevice(cudaDevice);
}

void CUDAEncryptHost(ubyte* input, ubyte* output, AesKey* encryptKey, int noBlocks)
{
	// Un bloc de fire de executie cripteaza 256 x 4 octeti = 1024 de octeti = 1K
	dim3 dimBlock(256);
	dim3 dimGrid(noBlocks); // Numarul de blocuri CUDA = numarul de blocuri de 1K din input

	// Alocam memorie pe device
	ubyte* deviceInput;
	cutilSafeCall(hipMalloc((void**)&deviceInput, noBlocks * 1024));

	ubyte* deviceOutput;
	cutilSafeCall(hipMalloc((void**)&deviceOutput, noBlocks * 1024));
	
	uint32* deviceRoundKeys;
	cutilSafeCall(hipMalloc((void**)&deviceRoundKeys, 4 * 4 * (AES_MAXNR + 1)));

	// Copiem input-ul pe device
	cutilSafeCall(hipMemcpy(deviceInput, input, noBlocks * 1024, hipMemcpyHostToDevice));

	// Copiem cheile
	cutilSafeCall(hipMemcpy(deviceRoundKeys, encryptKey->roundKeys,  
		4 * 4 * (AES_MAXNR + 1), hipMemcpyHostToDevice));

	// Cream un timer
    cutilCheckError(cutCreateTimer(&timer));
    cutilCheckError(cutStartTimer(timer));

	// Lansam kernelul
	CUDAEncryptKernel<<<dimGrid, dimBlock>>>(deviceInput, deviceOutput, 
		deviceRoundKeys, encryptKey->noRounds);

	// Verificam ca nu au aparut erori
	cutilCheckMsg("Kernel execution failed");

	// Copiem rezultatul de pe device
	cutilSafeCall(hipMemcpy(output, deviceOutput, noBlocks * 1024, hipMemcpyDeviceToHost));

	// Oprim timer-ul si afisam tipul de executie pe GPU
    cutilCheckError(cutStopTimer(timer));
    LogMessage("Processing time: %f (ms)", cutGetTimerValue(timer));
    cutilCheckError(cutDeleteTimer(timer));

	// Dealocam memoria pe device
	cutilSafeCall(hipFree(deviceInput));
    cutilSafeCall(hipFree(deviceOutput));
    cutilSafeCall(hipFree(deviceRoundKeys));

	// Oprim threaduri
	hipDeviceReset();
}

void CUDADecryptHost(ubyte* input, ubyte* output, AesKey* decryptKey, int noBlocks)
{
	// Un bloc de fire de executie cripteaza (16 x 16) x 4 octeti = 1024 de octeti = 1K
	dim3 dimBlock(256);	// 16 x 16 = un bloc de 4 x 4 octeti (o stare de 128 de biti) x 16 = 16 blocuri de 128
	dim3 dimGrid(noBlocks); // Numarul de blocuri CUDA = numarul de blocuri de 1K din input

	// Alocam memorie pe device
	ubyte* deviceInput;
	cutilSafeCall(hipMalloc((void**)&deviceInput, noBlocks * 1024));

	ubyte* deviceOutput;
	cutilSafeCall(hipMalloc((void**)&deviceOutput, noBlocks * 1024));
	
	uint32* deviceRoundKeys;
	cutilSafeCall(hipMalloc((void**)&deviceRoundKeys, 4 * 4 * (AES_MAXNR + 1)));

	// Copiem input-ul pe device
	cutilSafeCall(hipMemcpy(deviceInput, input, noBlocks * 1024, hipMemcpyHostToDevice));

	// Copiem cheile
	cutilSafeCall(hipMemcpy(deviceRoundKeys, decryptKey->roundKeys,  
		4 * 4 * (AES_MAXNR + 1), hipMemcpyHostToDevice));

	// Cream un timer
    cutilCheckError(cutCreateTimer(&timer));
    cutilCheckError(cutStartTimer(timer));

	// Lansam kernelul
	CUDADecryptKernel<<<dimGrid, dimBlock>>>(deviceInput, deviceOutput, 
		deviceRoundKeys, decryptKey->noRounds);

	// Verificam ca nu au aparut erori
	cutilCheckMsg("Kernel execution failed");

	// Copiem rezultatul de pe device
	cutilSafeCall(hipMemcpy(output, deviceOutput, noBlocks * 1024, hipMemcpyDeviceToHost));

	// Oprim timer-ul si afisam tipul de executie pe GPU
    cutilCheckError(cutStopTimer(timer));
    LogMessage("Processing time: %f (ms)", cutGetTimerValue(timer));
    cutilCheckError(cutDeleteTimer(timer));

	// Dealocam memoria pe device
	cutilSafeCall(hipFree(deviceInput));
    cutilSafeCall(hipFree(deviceOutput));
    cutilSafeCall(hipFree(deviceRoundKeys));

	// Oprim threaduri
	hipDeviceReset();
}

}